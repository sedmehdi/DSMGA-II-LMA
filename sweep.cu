#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2015 Tian-Li Yu and Shih-Huan Hsu                       *
 *   tianliyu@ntu.edu.tw                                                   *
 ***************************************************************************/

#include <cstdlib>
#include <cmath>
#include <iostream>
#include <fstream>

#include "statistics.h"
#include "dsmga2.cuh"
#include "global.cuh"
#define MAX_GEN 200

int step = 30;

using namespace std;

struct Record {
    int n;
    double nfe;
    double gen;
    double buildModelTime;
    double findCliqueTime;
};

__global__ void initCurandStates(hiprandState_t *d_randStates, int randSeed);

int main (int argc, char *argv[]) {

    if (argc != 4 && argc!=5 && argc !=6 && argc != 7) {
        printf ("sweep ell numConvergence function(0~3)\n");
        printf ("sweep ell numConvergence 4 [step #] [nk problem #]\n");
        printf ("sweep ell numConvergence 5 [spin problem #]\n");
        printf ("sweep ell numConvergence 6 [sat problem #]\n");
        printf ("function: \n");
        printf ("     ONEMAX:  0\n");
        printf ("     MK    :  1\n");
        printf ("     FTRAP :  2\n");
        printf ("     CYC   :  3\n");
        printf ("     NK    :  4\n");
        printf ("     SPIN  :  5\n");
        printf ("     SAT   :  6\n");
        return -1;
    }

    int ell = atoi (argv[1]);
    int numConvergence = atoi (argv[2]); // problem size
    int fffff = atoi(argv[3]);

    int problemNum = 0;
    int neighborNum = 0;
    int stepNum = 0;


    if (fffff == 4) {
        neighborNum = 4;
        stepNum = atoi (argv[4]);
        problemNum = atoi (argv[5]);
    }

    if (fffff == 5 || fffff == 6) {
        problemNum = atoi (argv[4]);
    }


    int nInitial = 40;
    hipMalloc(&d_randStates, sizeof(hiprandState_t) * ell * ell);
    initCurandStates<<<ell, ell>>>(d_randStates, 0);


    // for debug
    // myRand.seed(123);


    Statistics st;

    Statistics stGen, stNFE, stLS, stBuildModelTime, stFindCliqueTime;


    if (fffff == 5) {
	char filename[200];
        sprintf(filename, "./SPIN/%d/%d_%d",ell, ell, problemNum);
        if (SHOW_BISECTION) printf("Loading: %s\n", filename);
        loadSPIN(filename, &mySpinGlassParams);
    }

    if (fffff == 4) {
        char filename[200];
        sprintf(filename, "./NK_Instance/pnk%d_%d_%d_%d", ell, neighborNum, stepNum, problemNum);
        if (SHOW_BISECTION) printf("Loading: %s\n", filename);
        FILE *fp = fopen(filename, "r");
        loadNKWAProblem(fp, &nkwa);
        fclose(fp);
    }

    if (fffff == 6) {
        char filename[200];
        sprintf(filename, "./SAT/uf%d/uf%d-0%d.cnf",ell,ell,problemNum);
        if (SHOW_BISECTION) printf("Loading: %s\n", filename);
        loadSAT(filename, &mySAT);
    }


    bool foundOptima;
    Record rec[3];
    rec[0].n = nInitial;
    rec[1].n = nInitial+step;
    rec[2].n = nInitial+step+step;

    int popu;
    Record q1, q3;

    if (SHOW_BISECTION) printf("Bisection phase 1\n");

    for (int i=0; i<3; ++i) {
        popu = rec[i].n;

        if (SHOW_BISECTION) printf("[%d]: ", popu);

        foundOptima = true;

        stGen.reset();
        stNFE.reset();
        stLS.reset();
        stBuildModelTime.reset();
        stFindCliqueTime.reset();

        for (int j=0; j<numConvergence; j++) {

            DSMGA2 ga(ell, popu, MAX_GEN, -1, fffff, 0);
            ga.doIt(false);

            stGen.record(ga.getGeneration());
            stNFE.record(Chromosome::hitnfe);
            stLS.record(Chromosome::lsnfe);
            stBuildModelTime.record(ga.buildModelTime);
            stFindCliqueTime.record(ga.findCliqueTime);


            if (!ga.foundOptima()) {

                foundOptima = false;

                if (SHOW_BISECTION) {
                    printf("-");
                    fflush(NULL);
                }
                break;
            }

            if (SHOW_BISECTION) {
                printf("+");
                fflush(NULL);
            }
        }


        rec[i].gen = stGen.getMean();

        if (!foundOptima) {
            rec[i].nfe = INF;
            rec[i].buildModelTime = INF;
            rec[i].findCliqueTime = INF;
        }
        else {
            rec[i].nfe = stNFE.getMean();
            rec[i].buildModelTime = stBuildModelTime.getMean();
            rec[i].findCliqueTime = stFindCliqueTime.getMean();
        }
        if (SHOW_BISECTION) printf(" : %f, DSMTime: %f, ILSTime: %f\n", rec[i].nfe, rec[i].buildModelTime, rec[i].findCliqueTime);

    }

    while (rec[0].nfe < rec[1].nfe  && ((rec[2].n-rec[0].n)*20 > rec[1].n)) {

        rec[2] = rec[1];
        rec[1].n = (rec[0].n + rec[2].n) / 2;
        step /= 2;
        popu = rec[1].n;

        if (SHOW_BISECTION) printf("[%d]: ", popu);

        for (int j=0; j<numConvergence; j++) {

            DSMGA2 ga(ell, popu, MAX_GEN, -1, fffff, 0);
            ga.doIt(false);

            stGen.record(ga.getGeneration());
            stNFE.record(Chromosome::hitnfe);
            stLS.record(Chromosome::lsnfe);
            stBuildModelTime.record(ga.buildModelTime);
            stFindCliqueTime.record(ga.findCliqueTime);


            if (!ga.foundOptima()) {

                foundOptima = false;

                if (SHOW_BISECTION) {
                    printf("-");
                    fflush(NULL);
                }
                break;
            }

            if (SHOW_BISECTION) {
                printf("+");
                fflush(NULL);
            }
        }


        rec[1].gen = stGen.getMean();

        if (!foundOptima) {
            rec[1].nfe = INF;
            rec[1].buildModelTime = INF;
            rec[1].findCliqueTime = INF;
        }
        else {
            rec[1].nfe = stNFE.getMean();
            rec[1].buildModelTime = stBuildModelTime.getMean();
            rec[1].findCliqueTime = stFindCliqueTime.getMean();
        }
        if (SHOW_BISECTION) printf(" : %f, DSMTime: %f, ILSTime: %f\n", rec[1].nfe, rec[1].buildModelTime, rec[1].findCliqueTime);

    }


    while ( (rec[1].nfe >= rec[0].nfe) || (rec[1].nfe >= rec[2].nfe)) {

        popu = rec[2].n + step;

        if (SHOW_BISECTION) printf("[%d]: ", popu);

        foundOptima = true;

        stGen.reset();
        stNFE.reset();
        stLS.reset();
        stBuildModelTime.reset();
        stFindCliqueTime.reset();

        for (int j=0; j<numConvergence; j++) {

            DSMGA2 ga(ell, popu, MAX_GEN, -1, fffff, 0);
            ga.doIt(false);

            stGen.record(ga.getGeneration());
            stNFE.record(Chromosome::hitnfe);
            stLS.record(Chromosome::lsnfe);
            stBuildModelTime.record(ga.buildModelTime);
            stFindCliqueTime.record(ga.findCliqueTime);


            if (!ga.foundOptima()) {

                foundOptima = false;

                if (SHOW_BISECTION) {
                    printf("-");
                    fflush(NULL);
                }
                break;
            }

            if (SHOW_BISECTION) {
                printf("+");
                fflush(NULL);
            }
        }


        rec[0] = rec[1];
        rec[1] = rec[2];
        rec[2].n = popu;
        rec[2].gen = stGen.getMean();

        if (!foundOptima) {
            rec[2].nfe = INF;
            rec[2].buildModelTime = INF;
            rec[2].findCliqueTime = INF;
        }
        else {
            rec[2].nfe = stNFE.getMean();
            rec[2].buildModelTime = stBuildModelTime.getMean();
            rec[2].findCliqueTime = stFindCliqueTime.getMean();
        }
        if (SHOW_BISECTION) printf(" : %f, DSMTime: %f, ILSTime: %f\n", rec[2].nfe, rec[2].buildModelTime, rec[2].findCliqueTime);

    }


    if (SHOW_BISECTION) printf("Bisection phase 2\n");

    while ( ((rec[2].n-rec[0].n)*20 > rec[1].n) && (rec[2].n>rec[1].n+1) && (rec[1].n>rec[0].n+1)) {

        q1.n = (rec[0].n + rec[1].n) / 2;

        if (SHOW_BISECTION) printf("[%d]: ", q1.n);

        foundOptima = true;

        for (int j=0; j<numConvergence; j++) {

            DSMGA2 ga(ell, q1.n, MAX_GEN, -1, fffff, 0);
            ga.doIt(false);

            if (!ga.foundOptima()) {
                foundOptima = false;
                if (SHOW_BISECTION) {
                    printf("-");
                    fflush(NULL);
                }
                break;
            }
            if (SHOW_BISECTION) {
                printf("+");
                fflush(NULL);
            }
            if (j==0) {
                stGen.reset();
                stLS.reset();
                stNFE.reset();
                stBuildModelTime.reset();
                stFindCliqueTime.reset();
            }
            stGen.record(ga.getGeneration());
            stNFE.record(Chromosome::hitnfe);
            stLS.record(Chromosome::lsnfe);
            stBuildModelTime.record(ga.buildModelTime);
            stFindCliqueTime.record(ga.findCliqueTime);
        }

        q1.gen = stGen.getMean();
        if (foundOptima) {
            q1.nfe = stNFE.getMean();
            q1.buildModelTime = stBuildModelTime.getMean();
            q1.findCliqueTime = stFindCliqueTime.getMean();
        }
        else {
            q1.nfe = INF;
            q1.buildModelTime = INF;
            q1.findCliqueTime = INF;
        }

        if (SHOW_BISECTION) printf(" : %f, buildModelTime: %f, findCliqueTime: %f \n", q1.nfe, q1.buildModelTime, q1.findCliqueTime);


        q3.n = (rec[1].n + rec[2].n) / 2;

        if (SHOW_BISECTION) printf("[%d]: ", q3.n);

        foundOptima = true;

        for (int j=0; j<numConvergence; j++) {

            DSMGA2 ga(ell, q3.n, MAX_GEN, -1, fffff, 0);
            ga.doIt(false);

            if (!ga.foundOptima()) {
                foundOptima = false;
                if (SHOW_BISECTION) {
                    printf("-");
                    fflush(NULL);
                }
                break;
            }
            if (SHOW_BISECTION) {
                printf("+");
                fflush(NULL);
            }
            if (j==0) {
                stGen.reset();
                stLS.reset();
                stNFE.reset();
                stBuildModelTime.reset();
                stFindCliqueTime.reset();
            }
            stGen.record(ga.getGeneration());
            stNFE.record(Chromosome::hitnfe);
            stLS.record(Chromosome::lsnfe);
            stBuildModelTime.record(ga.buildModelTime);
            stFindCliqueTime.record(ga.findCliqueTime);
        }

        q3.gen = stGen.getMean();
        if (foundOptima) {
            q3.nfe = stNFE.getMean();
            q3.buildModelTime = stBuildModelTime.getMean();
            q3.findCliqueTime = stFindCliqueTime.getMean();
        } else {
            q3.nfe = INF;
            q3.buildModelTime = INF;
            q3.findCliqueTime = INF;
        }

        if (SHOW_BISECTION) printf(" : %f, buildModelTime: %f, findCliqueTime: %f \n", q3.nfe, q3. buildModelTime, q3.findCliqueTime);

        if (rec[1].nfe < q1.nfe && rec[1].nfe < q3.nfe) {
            rec[0] = q1;
            rec[2] = q3;
        } else if (q1.nfe < rec[1].nfe && q1.nfe < q3.nfe) {
            rec[2] = rec[1];
            rec[1] = q1;
        } else { // q3nfe smallest
            rec[0] = rec[1];
            rec[1] = q3;
        }
    };



    if (fffff == 4)
        freeNKWAProblem(&nkwa);

    printf("population: %d\n", rec[1].n);
    printf("generation: %f\n", rec[1].gen);
    printf("NFE: %f\n", rec[1].nfe);
    printf("buildModelTime: %f\n", rec[1].buildModelTime);
    printf("findCliqueTime: %f\n", rec[1].findCliqueTime);


    return EXIT_SUCCESS;

}

__global__ void initCurandStates(hiprandState_t *d_randStates, int randSeed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(randSeed, idx, 0, &d_randStates[idx]);
}
